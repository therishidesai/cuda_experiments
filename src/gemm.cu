
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdint.h>
#include <unistd.h>

#define N 16384
#define TILE_WIDTH 4

uint64_t nanos() {
  struct timespec start;
  clock_gettime(CLOCK_MONOTONIC_RAW, &start);
  return (uint64_t)start.tv_sec*1000000000 + (uint64_t)start.tv_nsec;
}

// Cd = Ad * Bd
__global__ void cuda_basic_gemm(float* Ad, float* Bd, float* Cd){
	float dp = 0;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (col >= N || row >= N) {
		return;
	}

	// do dot product of A row * B col
	for (int i = 0; i < N; i++) {
		dp += Ad[row * N + i] * Bd[i * N + col];
	}

	Cd[row * N + col] = dp;
}

// Tiled Cd = Ad * Bd
__global__ void cuda_shared_gemm(float *A, float *B, float *C) {

  __shared__ float tileM[TILE_WIDTH][TILE_WIDTH];
  __shared__ float tileN[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;
  int by = blockIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row = by * TILE_WIDTH + ty;
  int col = bx * TILE_WIDTH + tx;

  float result = 0;
  for(int t = 0; t < N/TILE_WIDTH; t++) {
    tileM[ty][tx] = A[row * N + t * TILE_WIDTH + tx];
    tileN[ty][tx] = B[(t * TILE_WIDTH + ty) * N + col];

    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; k++) {
      result += tileM[ty][k] * tileN[k][tx];
      __syncthreads();
    }

  }

  C[row * N + col] = result;
}

int main() {
    float* devA;
    float* devB;
    float* devC;
	float* A;
	float* B;
	float* C;
	float* val;

	// heap allocate for bigger matrices
	A = (float*) malloc(N * N * sizeof(float));
	B = (float*) malloc(N * N * sizeof(float));
	C = (float*) malloc(N * N * sizeof(float));
	val = (float*) malloc(N * N * sizeof(float));

	printf("Reading file\n");
    // Read matmul from numpy for validation
    // Took this from @geohot: https://github.com/geohot/tinygrad/blob/gemm/extra/gemm/gemm.c#L115
    FILE *f = fopen("/home/rishi/matmul", "rb");
    if (f == NULL) {
        printf("please pregenerate python /tmp/matmul file\n");
        return -1;
    }
    fread(A, 1, sizeof(float)*N*N, f);
    fread(B, 1, sizeof(float)*N*N, f);
    fread(val, 1, sizeof(float)*N*N, f);
    fclose(f);

	printf("CUDA!!!\n");

    hipMalloc((void**) &devA, N * N * sizeof(float));
    hipMalloc((void**) &devB, N * N * sizeof(float));
	hipMalloc((void**) &devC, N * N * sizeof(float));

	// Copy A and B to device memory
	hipMemcpy(devA, A, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devB, B, N * N * sizeof(float), hipMemcpyHostToDevice);

	// Thread block size of 32x32
	// Create a grid of 32x32 thread blocks
	dim3 dimGrid(ceil(N/32.0), ceil(N/32.0), 1);
	dim3 dimBlock(32, 32, 1);
	//dim3 dimGrid(ceil(N/double(TILE_WIDTH)), ceil(N/double(TILE_WIDTH)), 1);
	//dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

	uint64_t start = nanos();
    cuda_basic_gemm<<<dimGrid, dimBlock>>>(devA, devB, devC);
	//cuda_shared_gemm<<<dimGrid, dimBlock>>>(devA, devB, devC);
    hipDeviceSynchronize();

	// dumb CPU matmul
	/* for (int x = 0; x<N; x++) { */
	/* 	for (int y = 0; y<N; y++) { */
	/* 		C[x * N + y] = 0; */
	/* 		printf("C[%d]\n", x*N+y); */
	/* 		for (int k = 0; k < N; k++) { */
	/* 			printf("A[%d] * B[%d]\n", x*N+k, k*N+y); */
	/* 			C[x * N + y] += A[x * N + k] * B[k * N + y]; */
	/* 		} */
	/* 	} */
	/* } */
    uint64_t end = nanos();

	printf("GPU\n");
	double gflop = (2.0*N*N*N)*(1e-9);
	double s = (end-start)*(1e-9);
	printf("%f GFLOP\n", gflop);
	printf("%f sec\n", s);
	printf("%f GFLOP/S -- %.2f ms\n", gflop/s, s*1e3);

	hipMemcpy(C, devC, N * N * sizeof(float), hipMemcpyDeviceToHost);
	for (int k = 0; k < N*N; k++) {
		if (fabsf(C[k] - val[k]) > 1e-3) {
			printf("MISMATCH AT %d, %f != %f\n", k, C[k], val[k]);
			return -1;
		}
	}
	printf("match\n");
	

	hipFree(devA);
	hipFree(devB);
	hipFree(devC);

	free(A);
	free(B);
	free(C);
	free(val);
    return 0;
}
